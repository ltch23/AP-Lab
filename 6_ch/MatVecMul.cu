#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <iostream>

using namespace std;

__global__
void matVecMultKernel(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<n){
		C[i] = 0;
		for (int j = 0; j<n; j++)
			C[i] += A[j*n + i] * B[j];
	}
}

void matVecMult(float* A, float* B, float* C, int n) {
	int size = n * n * sizeof(float);
	int sizevect = n * sizeof(float);
	float *d_A, *d_B, *d_C;
	///Redimensionar y copiar de Host a Device
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_B, sizevect);
	hipMemcpy(d_B, B, sizevect, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, sizevect);

	//Llamada Kernel
	matVecMultKernel<<< ceil((n*n) / 256.0), 256 >>> (d_A, d_B, d_C, n);
	//ceil se asegura de que tener suficientes hilos para cubrir los elementos

	//copiar de Device a Host
	hipMemcpy(C, d_C, sizevect, hipMemcpyDeviceToHost);

	//liberar memoria
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

void showVecMult(float* matriz, float fila, float columna){
	
    for (int x = 0; x < fila; x++){
		for (int y = 0; y < columna; y++){
			int puesto = x*columna + y;
			printf("%3.0f ", matriz[puesto]);
		}
		printf("\n");
	}
}

int main() {

    int fila , columna ;
	cout<<"ingrese dimensiones"<< endl;
	cin>>fila;
	columna=fila;
	float* A = (float*)malloc(fila*columna*sizeof(float));
	float* B = (float*)malloc(fila*sizeof(float));
	float* C = (float*)malloc(fila*sizeof(float));
	for (int i = 0; i < fila*columna; i++)
        A[i] = i;
    for (int i = 0; i < columna; i++)
        B[i] = i;
	
    cout<<" vector "<<endl;
	showVecMult(B, 1, columna);
	cout<<" * matriz "<<endl;
	showVecMult(A, fila, columna);
	
	cout <<"Resultado"<<endl;
	matrizXvector(A, B, C, fila);
	showVecMult(C, 1, columna);

	system("PAUSE");
	exit(0);
}
